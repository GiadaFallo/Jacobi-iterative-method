#include "hip/hip_runtime.h"


#include<stdio.h>
#include<iostream>
#include<stdlib.h>
#include<getopt.h>
#include <assert.h>

#include <chrono>

#define RAND_RANGE_MIN -10.0
#define RAND_RANGE_MAX 10.0
#define SEED 123
#define JACOBI_DEBUG 0

enum ERROR_TYPE { MEMCPY, CMALLOC, ADDK, DEVSYNC };

void init_const(float* v, float x, int dim) {
  for (int i = 0; i < dim; i++)
    v[i] = x;
  return;
}

void init_matrix(float **A, int matrix_order) {

  for (int i = 0; i < matrix_order; i++) {
    A[i] = new float[matrix_order];
    if (A[i] == NULL) {
      std::cerr << "Error while allocating resources." << std::endl;
      exit(-1);
    }
  }
}

/** Generates a random number in a specific range.

    @param fMin The lower bound of the range.
    @param fMax The upper bound of the range.
    @return The generated number.
*/
float generate_random_number(float fMin, float fMax) {
  float f = (float)rand() / RAND_MAX;

  return fMin + f * (fMax - fMin);
}

/** Generates a random square matrix.
    @param A The matrix.
*/
void generate_random_matrix(float **A, int matrix_order) {

  for (int i = 0; i < matrix_order; i++) {
    float sum = 0.0;
    for (int j = 0; j < matrix_order; j++)
      if (j != i) {
        float val = generate_random_number(RAND_RANGE_MIN, RAND_RANGE_MAX);
        sum += abs(val);

        A[i][j] = val;
      }

    /* Change back A[i][i] to be > then sum(A[i][j]) */
    A[i][i] = sum + generate_random_number(1.0, RAND_RANGE_MAX);
  }
}

/** Generates a random vector.
    @param v .
*/
void generate_random_vector(float *v, int matrix_order) {
  /* generate vector v */
  for (int j = 0; j < matrix_order; j++) {
    float val = generate_random_number(RAND_RANGE_MIN, RAND_RANGE_MAX);
    v[j] = val;
  }
}

/** Generate a random number in a specific range.
    @param A The square matrix.
    @param v The vector.
    @param start .
    @param end .
*/
void matrix_vector_multiplication(float *x, float **A, float *v, int matrix_order) {

  for (int i = 0; i < matrix_order; i++) {
    x[i] = 0;
    for (int j = 0; j < matrix_order; j++)
      x[i] += A[i][j] * v[j];
  }
  return;
}

void error_on_computation(float* x, float ** A, float *b, int matrix_order, float *err) {
    float error = 0.0, sum = 0.0;
  
    for (size_t i = 0; i < matrix_order; i++) {
      sum = 0.0;
      for (size_t j = 0; j < matrix_order; j++) {
          
        sum = sum + A[i][j] * x[j]; 
      }
      error = error + abs(sum - b[i]);
    }
    *err = error / matrix_order;
    return;
}


std::chrono::duration<double> delta_time(std::chrono::time_point<std::chrono::system_clock> start, std::chrono::time_point<std::chrono::system_clock> end) {
  return end - start;
}


hipError_t error_check(hipError_t cudaStatus, ERROR_TYPE msgtype, float*dev_a, float*dev_x_solution, float*dev_b, float*dev_prec_values){
	
    if (cudaStatus != hipSuccess) {

        switch(msgtype) {
            case (CMALLOC):{
                std::cerr <<  "hipMalloc failed!" << std::endl;
            }
            case (MEMCPY):{
                std::cerr <<  "hipMemcpy failed!" << std::endl;
            }
            case (ADDK):{
                std::cerr << "addKernel launch failed:" << hipGetErrorString(cudaStatus) << std::endl;
            }
            case(DEVSYNC):{
                std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus <<  " after launching jacobi!"  << std::endl;
            }
        }
		
    hipFree(dev_a);
    hipFree(dev_x_solution);
    hipFree(dev_prec_values);
    hipFree(dev_b);
	
    return cudaStatus;
	}
}

__global__ void iteration(float * a, float * x_solution, float * b, float * prec_values, unsigned int matrix_order) { 
    unsigned int j, i;
	float sigma = 0.0, newValue = 0.0;
    int bx = blockIdx.x, tx = threadIdx.x;
    i = tx + bx*blockDim.x;
    
    if (i >= matrix_order) return;

    if (i < matrix_order){
        sigma = b[i];
        int idx_Ai = i*matrix_order;
        
        for (j = 0; j < matrix_order; j++) {
            if (i != j) { sigma = sigma - a[idx_Ai + j] * x_solution[j]; }
        }

        newValue = sigma / a[idx_Ai + i];
        
        prec_values[i] = (x_solution[i] - newValue)*(x_solution[i] - newValue);
        x_solution[i] = newValue;
        __syncthreads();
    }
}

hipError_t cuda_jacobi_solve(float * a, float * x_solution, float * b, float eps, unsigned int matrix_order, int * max_iter, float *prec) {
	unsigned int i, j;
	
    int k = 0, nTiles;
	float *dev_a = 0, *dev_x_solution = 0, *dev_b = 0, *dev_prec_values = 0;
    float accur = 1.0, sum = 0.0;

    float *prec_values = new float[matrix_order];
    init_const(prec_values, 0.0, matrix_order);

    size_t matrix_size = matrix_order*matrix_order*sizeof(float);
    size_t vector_size = matrix_order*sizeof(float);

	hipError_t cudaStatus;

    hipDeviceSetCacheConfig(hipFuncCachePreferShared);

	cudaStatus = hipMalloc((void**)&dev_a, matrix_size);
    error_check(cudaStatus, CMALLOC, dev_a, dev_x_solution, dev_b, dev_prec_values);

	cudaStatus = hipMalloc((void**)&dev_x_solution, vector_size);
    error_check(cudaStatus, CMALLOC, dev_a, dev_x_solution, dev_b, dev_prec_values);

	cudaStatus = hipMalloc((void**)&dev_b, vector_size);
    error_check(cudaStatus, CMALLOC, dev_a, dev_x_solution, dev_b, dev_prec_values);

    cudaStatus = hipMalloc((void**)&dev_prec_values, vector_size);
    error_check(cudaStatus, CMALLOC, dev_a, dev_x_solution, dev_b, dev_prec_values); 

	cudaStatus = hipMemcpy(dev_a, a, matrix_size, hipMemcpyHostToDevice);
    error_check(cudaStatus, MEMCPY, dev_a, dev_x_solution, dev_b, dev_prec_values);

	cudaStatus = hipMemcpy(dev_x_solution, x_solution, vector_size, hipMemcpyHostToDevice);
    error_check(cudaStatus, MEMCPY, dev_a, dev_x_solution, dev_b, dev_prec_values);

	cudaStatus = hipMemcpy(dev_b, b, vector_size, hipMemcpyHostToDevice);
    error_check(cudaStatus, MEMCPY, dev_a, dev_x_solution, dev_b, dev_prec_values);

	cudaStatus = hipMemcpy(dev_prec_values, prec_values, vector_size, hipMemcpyHostToDevice);    
    error_check(cudaStatus, MEMCPY, dev_a, dev_x_solution, dev_b, dev_prec_values);    

    int tileSize = 16;
    nTiles = matrix_order/tileSize + (matrix_order%tileSize == 0?0:1);

    for (i = 0; i < *max_iter; i++) {

        iteration <<<nTiles,tileSize>>> (dev_a, dev_x_solution, dev_b, dev_prec_values, matrix_order);
        k++;

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        error_check(cudaStatus, ADDK, dev_a, dev_x_solution, dev_b, dev_prec_values);

        // cudaDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        error_check(cudaStatus, DEVSYNC, dev_a, dev_x_solution, dev_b, dev_prec_values);

        // Retreive the dev_prec_values vector with all the precision values
        cudaStatus = hipMemcpy(prec_values, dev_prec_values, vector_size, hipMemcpyDeviceToHost);
        error_check(cudaStatus, MEMCPY, dev_a, dev_x_solution, dev_b, dev_prec_values);    

        // Computes the precision 
        sum = 0.0;
        for (j = 0; j < matrix_order; j++) {
            sum = sum + fabs(prec_values[j]);
        }
        accur = sqrt(sum);
        if (accur <= eps) break;
	}

	*max_iter = k;
    *prec = accur;
	cudaStatus = hipMemcpy(x_solution, dev_x_solution, vector_size, hipMemcpyDeviceToHost);
    error_check(cudaStatus, MEMCPY, dev_a, dev_x_solution, dev_b, dev_prec_values); 

    hipFree(dev_a);
    hipFree(dev_x_solution);
    hipFree(dev_prec_values);
    hipFree(dev_b);
}

int main(int argc, char *argv[]){

    const int matrix_order = atoi(argv[1]); // order of the matrix
    int max_iter = atoi(argv[2]);     // number of max_iterations
    const float epsilon = atof(argv[3]);    // precision

    int iterations = max_iter;
    std::chrono::time_point<std::chrono::system_clock> start_time, end_time;

    float *x_solution_h, *b_h, **A_h, *rand_x_h;
    float *extended_a = 0;    

    // Allocate memory for CPU.
    A_h = new float *[matrix_order];
    b_h = new float[matrix_order];
    x_solution_h = new float[matrix_order];
    rand_x_h = new float[matrix_order];

    if (A_h == NULL || b_h == NULL || rand_x_h == NULL || x_solution_h == NULL) {
        std::cerr << "Error while allocating resources." << std::endl;
        exit(-1);
    }

    init_matrix(A_h, matrix_order);
    srand(SEED);
    generate_random_matrix(A_h, matrix_order);
    
    extended_a = (float*)malloc(matrix_order*matrix_order*sizeof(float));
	
    for (int i = 0; i < matrix_order; i++) {
		for (int j = 0; j < matrix_order; j++) {
			extended_a[i*matrix_order + j] = A_h[i][j];
		}
	}
    
    generate_random_vector(rand_x_h, matrix_order);

    int repetitions = 20;
    float precision = 1.0, err = 0.0;
    
    matrix_vector_multiplication(b_h, A_h, rand_x_h, matrix_order);
    
    for (int m=0; m<repetitions; m++) {
        
        init_const(x_solution_h, 0.0, matrix_order);
        iterations = max_iter;
        err=0.0;

        start_time = std::chrono::system_clock::now();
        cuda_jacobi_solve(extended_a, x_solution_h, b_h, epsilon, matrix_order, &iterations, &precision);
        end_time = std::chrono::system_clock::now();

        std::cout << delta_time(start_time, end_time).count()  << "\t" ;        
        error_on_computation(x_solution_h, A_h, b_h, matrix_order, &err);
    }

        std::cout << "\t" << iterations << "\t" <<  precision << "\t" <<  err  << std::endl;            

    // Release resources
    for (int i = 0; i < matrix_order; i++)
        delete[] A_h[i];

    delete[] A_h;
    delete[] b_h;
    delete[] rand_x_h;
    delete[] x_solution_h;
    free(extended_a);

    return 0;

}